// actual_test_cuda.cu
// Usage: ./actual_test_cuda <data_size> <axis> <iter_count> <iter_batch> <run_count>
// Output: fft_cuda_<axis>_axis.csv with the same columns as your Torch script.
//
// Build (example):
//   nvcc -O3 -std=c++17 actual_test_cuda.cu -lcufft -o actual_test_cuda

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <string>
#include <vector>
#include <cmath>

static inline void checkCuda(hipError_t err, const char* what) {
    if (err != hipSuccess) {
        std::cerr << "[CUDA] " << what << " failed: " << hipGetErrorString(err) << "\n";
        std::exit(1);
    }
}

static inline void checkCuFFT(hipfftResult err, const char* what) {
    if (err != HIPFFT_SUCCESS) {
        std::cerr << "[cuFFT] " << what << " failed: " << err << "\n";
        std::exit(1);
    }
}

struct Config {
    long long data_size;
    int axis;          // 0 or 1
    int iter_count;
    int iter_batch;
    int run_count;
    int warmup = 10;   // match Torch script’s warmup
};

static Config parse_args(int argc, char** argv) {
    if (argc != 6) {
        std::cerr << "Usage: " << argv[0]
                  << " <data_size> <axis> <iter_count> <iter_batch> <run_count>\n";
        std::exit(1);
    }
    Config c;
    c.data_size  = std::stoll(argv[1]);
    c.axis       = std::stoi(argv[2]);
    c.iter_count = std::stoi(argv[3]);
    c.iter_batch = std::stoi(argv[4]);
    c.run_count  = std::stoi(argv[5]);
    if (c.axis != 0 && c.axis != 1) {
        std::cerr << "axis must be 0 or 1\n";
        std::exit(1);
    }
    return c;
}

static std::vector<int> get_fft_sizes() {
    std::vector<int> sizes;
    for (int p = 6; p <= 12; ++p) sizes.push_back(1 << p); // 64..4096
    return sizes;
}

// Compute GB processed per single FFT execution (read + write) for shape (dim0, dim1)
static double gb_per_exec(long long dim0, long long dim1) {
    // complex64 = 8 bytes; count both read and write -> *2
    const double bytes = 2.0 * static_cast<double>(dim0) * static_cast<double>(dim1) * 8.0;
    return bytes / (1024.0 * 1024.0 * 1024.0);
}

static double run_cufft_case(const Config& cfg, int fft_size) {
    // Shape has two dims; size along 'axis' is fft_size, the other is data_size / fft_size
    const int batched_axis = (cfg.axis + 1) % 2;

    long long dims[2] = {0, 0};
    dims[cfg.axis] = fft_size;
    dims[batched_axis] = cfg.data_size / fft_size;

    if (dims[batched_axis] <= 0) {
        // Nothing to do (mismatch), return 0
        return 0.0;
    }

    const long long dim0 = dims[0];
    const long long dim1 = dims[1];
    const long long total_elems = dim0 * dim1;

    // Device buffers (in-place transform will overwrite input)
    hipfftComplex* d_data = nullptr;
    checkCuda(hipMalloc(&d_data, total_elems * sizeof(hipfftComplex)), "hipMalloc d_data");
    // Optionally zero-fill
    checkCuda(hipMemset(d_data, 0, total_elems * sizeof(hipfftComplex)), "hipMemset d_data");

    // cuFFT plan: 1D transforms along 'axis', batched over the other dimension
    hipfftHandle plan;
    checkCuFFT(hipfftCreate(&plan), "hipfftCreate");

    int n[1] = { fft_size }; // 1D length

    // Strides and distances depend on axis in row-major layout:
    // Layout: index (i0, i1) -> offset = i0*dim1 + i1
    int istride, ostride, idist, odist, batch;
    int inembed[1] = {0};  // not used when rank=1 unless advanced layouts; set 0
    int onembed[1] = {0};

    if (cfg.axis == 1) {
        // Transform along contiguous last dimension (dim1)
        // Each transform is a row of length dim1; there are dim0 batches.
        istride = 1;
        ostride = 1;
        idist   = static_cast<int>(dim1);
        odist   = static_cast<int>(dim1);
        batch   = static_cast<int>(dim0);
    } else {
        // Transform along first dimension (dim0)
        // Elements of a single transform are separated by dim1
        istride = static_cast<int>(dim1);
        ostride = static_cast<int>(dim1);
        idist   = 1;
        odist   = 1;
        batch   = static_cast<int>(dim1);
    }

    // Create plan (complex-to-complex, single-precision)
    checkCuFFT(
        hipfftPlanMany(&plan,
                      /*rank*/ 1,
                      n,
                      inembed,  istride, idist,
                      onembed,  ostride, odist,
                      HIPFFT_C2C,
                      batch),
        "hipfftPlanMany"
    );

    // Warmup
    for (int i = 0; i < cfg.warmup; ++i) {
        checkCuFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD), "hipfftExecC2C warmup");
    }
    checkCuda(hipDeviceSynchronize(), "hipDeviceSynchronize (after warmup)");

    // Time iter_count * iter_batch executions
    hipEvent_t ev_start, ev_stop;
    checkCuda(hipEventCreate(&ev_start), "hipEventCreate start");
    checkCuda(hipEventCreate(&ev_stop),  "hipEventCreate stop");
    checkCuda(hipEventRecord(ev_start),  "hipEventRecord start");

    for (int it = 0; it < cfg.iter_count; ++it) {
        checkCuFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD), "hipfftExecC2C timed");
    }

    checkCuda(hipEventRecord(ev_stop), "hipEventRecord stop");
    checkCuda(hipEventSynchronize(ev_stop), "hipEventSynchronize stop");

    float ms = 0.0f;
    checkCuda(hipEventElapsedTime(&ms, ev_start, ev_stop), "hipEventElapsedTime");

    // Cleanup events (keep data around until after sync)
    hipEventDestroy(ev_start);
    hipEventDestroy(ev_stop);

    // Convert elapsed to seconds
    const double seconds = static_cast<double>(ms) / 1000.0;

    // Compute throughput in GB/s (same accounting as Torch: 2 * elems * 8 bytes per exec)
    const double gb_per_exec_once = gb_per_exec(dim0, dim1);
    const double total_execs = static_cast<double>(cfg.iter_count); // * static_cast<double>(cfg.iter_batch);
    const double gb_per_second = (total_execs * gb_per_exec_once) / seconds;

    // Cleanup
    hipfftDestroy(plan);
    hipFree(d_data);

    return gb_per_second;
}

int main(int argc, char** argv) {
    const Config cfg = parse_args(argc, argv);
    const auto sizes = get_fft_sizes();

    const std::string output_name = "fft_cuda_" + std::to_string(cfg.axis) + "_axis.csv";
    std::ofstream out(output_name);
    if (!out) {
        std::cerr << "Failed to open output file: " << output_name << "\n";
        return 1;
    }

    std::cout << "Running cuFFT tests with data size " << cfg.data_size
              << ", axis " << cfg.axis
              << ", iter_count " << cfg.iter_count
              << ", iter_batch " << cfg.iter_batch
              << ", run_count " << cfg.run_count << "\n";

    // Header: Backend, FFT Size, Run 1..N, Mean, Std Dev
    out << "Backend,FFT Size";
    for (int i = 0; i < cfg.run_count; ++i) out << ",Run " << (i + 1) << " (GB/s)";
    out << ",Mean,Std Dev\n";

    for (int fft_size : sizes) {
        std::vector<double> rates;
        rates.reserve(cfg.run_count);

        for (int r = 0; r < cfg.run_count; ++r) {
            const double gbps = run_cufft_case(cfg, fft_size);
            std::cout << "FFT Size: " << fft_size << ", Throughput: " << std::fixed << std::setprecision(2)
                      << gbps << " GB/s\n";
            rates.push_back(gbps);
        }

        // Compute mean/std
        double mean = 0.0;
        for (double v : rates) mean += v;
        mean /= static_cast<double>(rates.size());

        double var = 0.0;
        for (double v : rates) {
            const double d = v - mean;
            var += d * d;
        }
        var /= static_cast<double>(rates.size());
        const double stdev = std::sqrt(var);

        // Round to 2 decimals like your Torch script
        out << "cuda," << fft_size;
        out << std::fixed << std::setprecision(2);
        for (double v : rates) out << "," << v;
        out << "," << mean << "," << stdev << "\n";
    }

    std::cout << "Results saved to " << output_name << "\n";
    return 0;
}
