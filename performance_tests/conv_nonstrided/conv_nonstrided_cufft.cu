// actual_test_cuda.cu
// Usage: ./actual_test_cuda <data_size> <axis> <iter_count> <iter_batch> <run_count>
// Output: fft_cuda_<axis>_axis.csv with the same columns as your Torch script.
//
// Build (example):
//   nvcc -O3 -std=c++17 actual_test_cuda.cu -lcufft -o actual_test_cuda

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <string>
#include <vector>
#include <cmath>

__global__ void fill_randomish(hipfftComplex* a, long long n){
    long long i = blockIdx.x * 1LL * blockDim.x + threadIdx.x;
    if(i<n){
        float x = __sinf(i * 0.00173f);
        float y = __cosf(i * 0.00091f);
        a[i] = make_float2(x, y);
    }
}

__global__ void scale_kernel(hipfftComplex* data, float scale_factor, long long total_elems) {
    long long i = blockIdx.x * 1LL * blockDim.x + threadIdx.x;
    if (i < total_elems) {
        data[i].x *= scale_factor;
        data[i].y *= scale_factor;
    }
}

static inline void checkCuda(hipError_t err, const char* what) {
    if (err != hipSuccess) {
        std::cerr << "[CUDA] " << what << " failed: " << hipGetErrorString(err) << "\n";
        std::exit(1);
    }
}

static inline void checkCuFFT(hipfftResult err, const char* what) {
    if (err != HIPFFT_SUCCESS) {
        std::cerr << "[cuFFT] " << what << " failed: " << err << "\n";
        std::exit(1);
    }
}

struct Config {
    long long data_size;
    int iter_count;
    int iter_batch;
    int run_count;
    int warmup = 10;   // match Torch script’s warmup
};

static Config parse_args(int argc, char** argv) {
    if (argc != 5) {
        std::cerr << "Usage: " << argv[0]
                  << " <data_size> <iter_count> <iter_batch> <run_count>\n";
        std::exit(1);
    }
    Config c;
    c.data_size  = std::stoll(argv[1]);
    c.iter_count = std::stoi(argv[2]);
    c.iter_batch = std::stoi(argv[3]);
    c.run_count  = std::stoi(argv[4]);
    return c;
}

static std::vector<int> get_fft_sizes() {
    std::vector<int> sizes;
    for (int p = 6; p <= 12; ++p) sizes.push_back(1 << p); // 64..4096
    return sizes;
}

// Compute GB processed per single FFT execution (read + write) for shape (dim0, dim1)
static double gb_per_exec(long long dim0, long long dim1) {
    // complex64 = 8 bytes; count both read and write -> *2
    const double bytes = static_cast<double>(dim0) * static_cast<double>(dim1) * 8.0;
    return bytes / (1024.0 * 1024.0 * 1024.0);
}

static double run_cufft_case(const Config& cfg, int fft_size) {
    const long long dim0 = cfg.data_size / fft_size;
    const long long dim1 = fft_size;
    const long long total_elems = dim0 * dim1;

    // Device buffers (in-place transform will overwrite input)
    hipfftComplex* d_data = nullptr;
    checkCuda(hipMalloc(&d_data, total_elems * sizeof(hipfftComplex)), "hipMalloc d_data");
    // Optionally zero-fill
    checkCuda(hipMemset(d_data, 0, total_elems * sizeof(hipfftComplex)), "hipMemset d_data");

    hipfftComplex* d_kernel = nullptr;
    checkCuda(hipMalloc(&d_kernel, (total_elems) * sizeof(hipfftComplex)), "hipMalloc d_kernel");
    // Optionally zero-fill
    checkCuda(hipMemset(d_kernel, 0, (total_elems) * sizeof(hipfftComplex)), "hipMemset d_kernel");

    {
        int t = 256, b = int((total_elems + t - 1) / t);
        fill_randomish<<<b,t>>>(d_data, total_elems);
        checkCuda(hipGetLastError(), "fill launch");
        checkCuda(hipDeviceSynchronize(), "fill sync");

        int kt = 256, kb = int((total_elems + kt - 1) / kt);
        fill_randomish<<<kb,kt>>>(d_kernel, total_elems);
        checkCuda(hipGetLastError(), "fill kernel launch");
        checkCuda(hipDeviceSynchronize(), "fill kernel sync");
    }

    // --- plan bound to the stream ---
    hipfftHandle plan;
    checkCuFFT(hipfftCreate(&plan), "hipfftCreate");

    // int n[2] = { int(dim1), int(dim2) };
    // int inembed[2] = { int(dim1), int(dim2) };        // physical layout (same as n for tight pack)
    // int onembed[2] = { int(dim1), int(dim2) };
    // int istride    = 1;               // contiguous within each 2D image
    // int ostride    = 1;
    // int idist      = int(dim1)* int(dim2);           // distance between images
    // int odist      = int(dim1)* int(dim2);

    // checkCuFFT(cufftPlanMany(&plan, 2, n,
    //                               inembed,  istride, idist,
    //                               onembed,  ostride, odist,
    //                               CUFFT_C2C, int(dim0)), "plan2d");

    checkCuFFT(hipfftPlan1d(&plan, dim1, HIPFFT_C2C, dim0), "plan");

    // --- warmup on the stream ---
    for (int i = 0; i < cfg.warmup; ++i) {
        checkCuFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD), "warmup");
        scale_kernel<<<(total_elems+255)/256,256>>>(d_data, 5.0, total_elems);
        checkCuFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD), "warmup");
    }
    
    checkCuda(hipDeviceSynchronize(), "warmup sync");

    // === OPTION A: plain single-stream timing (simple & robust) ===
    hipEvent_t evA, evB;
    checkCuda(hipEventCreate(&evA), "evA");
    checkCuda(hipEventCreate(&evB), "evB");
    checkCuda(hipEventRecord(evA), "record A");
    for (int it = 0; it < cfg.iter_count; ++it) {
        checkCuFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD), "exec");
        scale_kernel<<<(total_elems+255)/256,256>>>(d_data, 5.0, total_elems);
        checkCuFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD), "exec");
    }
    checkCuda(hipEventRecord(evB), "record B");
    checkCuda(hipEventSynchronize(evB), "sync B");
    checkCuda(hipDeviceSynchronize(), "warmup sync");
    float ms = 0.f; checkCuda(hipEventElapsedTime(&ms, evA, evB), "elapsed");
    checkCuda(hipEventDestroy(evA), "dA");
    checkCuda(hipEventDestroy(evB), "dB");

    // Convert elapsed to seconds
    const double seconds = static_cast<double>(ms) / 1000.0;

    // Compute throughput in GB/s (same accounting as Torch: 2 * elems * 8 bytes per exec)
    const double gb_per_exec_once = 6 * gb_per_exec(dim0, dim1);
    const double total_execs = static_cast<double>(cfg.iter_count); // * static_cast<double>(cfg.iter_batch);
    const double gb_per_second = (total_execs * gb_per_exec_once) / seconds;

    // Cleanup
    hipfftDestroy(plan);
    hipFree(d_data);
    hipFree(d_kernel);

    return gb_per_second;
}

int main(int argc, char** argv) {
    const Config cfg = parse_args(argc, argv);
    const auto sizes = get_fft_sizes();

    const std::string output_name = "conv_nonstrided_cufft.csv";
    std::ofstream out(output_name);
    if (!out) {
        std::cerr << "Failed to open output file: " << output_name << "\n";
        return 1;
    }

    std::cout << "Running cuFFT tests with data size " << cfg.data_size
              << ", iter_count " << cfg.iter_count
              << ", iter_batch " << cfg.iter_batch
              << ", run_count " << cfg.run_count << "\n";

    // Header: Backend, FFT Size, Run 1..N, Mean, Std Dev
    out << "Backend,FFT Size";
    for (int i = 0; i < cfg.run_count; ++i) out << ",Run " << (i + 1) << " (GB/s)";
    out << ",Mean,Std Dev\n";

    for (int fft_size : sizes) {
        std::vector<double> rates;
        rates.reserve(cfg.run_count);

        for (int r = 0; r < cfg.run_count; ++r) {
            const double gbps = run_cufft_case(cfg, fft_size);
            std::cout << "FFT Size: " << fft_size << ", Throughput: " << std::fixed << std::setprecision(2)
                      << gbps << " GB/s\n";
            rates.push_back(gbps);
        }

        // Compute mean/std
        double mean = 0.0;
        for (double v : rates) mean += v;
        mean /= static_cast<double>(rates.size());

        double var = 0.0;
        for (double v : rates) {
            const double d = v - mean;
            var += d * d;
        }
        var /= static_cast<double>(rates.size());
        const double stdev = std::sqrt(var);

        // Round to 2 decimals like your Torch script
        out << "cufft," << fft_size;
        out << std::fixed << std::setprecision(2);
        for (double v : rates) out << "," << v;
        out << "," << mean << "," << stdev << "\n";
    }

    std::cout << "Results saved to " << output_name << "\n";
    return 0;
}
