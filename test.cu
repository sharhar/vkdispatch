#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <cstdio>
#include <cstdlib>
#include <chrono>

#define CHECK_CUDA(call) do { \
    hipError_t err = call;   \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE);   \
    }                         \
} while (0)

#define CHECK_CUFFT(call) do { \
    hipfftResult err = call;    \
    if (err != HIPFFT_SUCCESS) { \
        fprintf(stderr, "cuFFT error %s:%d: %d\n", __FILE__, __LINE__, err); \
        exit(EXIT_FAILURE);    \
    }                          \
} while (0)

int main() {
    // FFT parameters
    const int NX = 4096;
    const int NY = 4096;
    const int BATCH = 10;         // Number of images in batch (adjust as desired)
    const int NUM_ITER = 1000;    // Number of forward+inverse pairs

    size_t total_elems = size_t(NX) * NY * BATCH;

    // Host memory for initialization
    hipfftComplex *h_input = (hipfftComplex*)malloc(total_elems * sizeof(hipfftComplex));
    for (size_t i = 0; i < total_elems; ++i) {
        h_input[i].x = (float)rand() / RAND_MAX;
        h_input[i].y = (float)rand() / RAND_MAX;
    }

    // Device memory
    hipfftComplex *d_data;
    CHECK_CUDA(hipMalloc(&d_data, total_elems * sizeof(hipfftComplex)));
    CHECK_CUDA(hipMemcpy(d_data, h_input, total_elems * sizeof(hipfftComplex), hipMemcpyHostToDevice));

    // cuFFT plan (batched 2D C2C)
    hipfftHandle plan;
    int n[2] = {NX, NY};
    int inembed[2] = {NX, NY};
    int onembed[2] = {NX, NY};
    int istride = 1, ostride = 1;
    int idist = NX * NY, odist = NX * NY;
    CHECK_CUFFT(hipfftPlanMany(&plan, 2, n,
                              inembed, istride, idist,
                              onembed, ostride, odist,
                              HIPFFT_C2C, BATCH));

    // Warmup
    CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD));
    CHECK_CUDA(hipDeviceSynchronize());

    // Timing
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < NUM_ITER; ++i) {
        CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));
        CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD));
    }

    CHECK_CUDA(hipMemcpy(h_input, d_data, total_elems * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();

    // Report
    double elapsed = std::chrono::duration<double>(end - start).count();
    double pairs_per_sec = (double)NUM_ITER * BATCH / elapsed;
    printf("%d x %d x %d C2C 2D FFTs: %.2f forward+inverse pairs per second (total: %d cycles in %.3f s)\n",
           NX, NY, BATCH, pairs_per_sec, NUM_ITER, elapsed);

    // Cleanup
    hipfftDestroy(plan);
    hipFree(d_data);
    free(h_input);
    return 0;
}
